#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "DS_timer.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

// The size of the vector
#define NUM_DATA 1024 * 1024

// Simple vector sum kernel
__global__ void vecAdd(int* _a, int* _b, int* _c, int _size) {
    int tID = blockIdx.x * blockDim.x + threadIdx.x;
    if (tID < _size)
        _c[tID] = _a[tID] + _b[tID];
}

int main(void)
{
    int* a,  * b, * c, * hc;    // Vectors on the host
    int* da, * db, * dc;        // Vector on the device

    dim3 dimGrid(ceil((float)NUM_DATA / 256), 1, 1);
    dim3 dimBlock(256, 1, 1);

    int memSize = sizeof(int) * NUM_DATA;
    printf("%d elements, memSize = %d bytes\n", NUM_DATA, memSize);

    DS_timer timer(5);
    timer.initTimers();

    // Memory allocation on the host-side
    a = new int[NUM_DATA]; memset(a, 0, memSize);
    b = new int[NUM_DATA]; memset(b, 0, memSize);
    c = new int[NUM_DATA]; memset(c, 0, memSize);
    hc = new int[NUM_DATA]; memset(hc, 0, memSize);

    // Data generation
    for (int i = 0; i < NUM_DATA; i++) {
        a[i] = rand() % 10;
        b[i] = rand() % 10;
    }

    // Vector sum on host (for performance comparsion)
    timer.onTimer(0);
    for (int i = 0; i < NUM_DATA; i++)
        hc[i] = a[i] + b[i];
    timer.offTimer(0);

    // Memory allocation on the device-side
    hipMalloc(&da, memSize); hipMemset(da, 0, memSize);
    hipMalloc(&db, memSize); hipMemset(db, 0, memSize);
    hipMalloc(&dc, memSize); hipMemset(dc, 0, memSize);
    
    timer.onTimer(2);
    // Data copy : Host -> Device
    hipMemcpy(da, a, memSize, hipMemcpyHostToDevice);
    hipMemcpy(db, b, memSize, hipMemcpyHostToDevice);
    timer.offTimer(2);

    // Kernel call
    timer.onTimer(1);
    vecAdd <<<dimGrid, dimBlock>>> (da, db, dc, NUM_DATA);
    hipDeviceSynchronize();
    timer.offTimer(1);

    // Copy results: Device -> Host
    hipMemcpy(c, dc, memSize, hipMemcpyDeviceToHost);

    // Release device memory
    hipFree(da); hipFree(db); hipFree(dc);

    // Check results
    bool result = true;
    for (int i = 0; i < NUM_DATA ; i++) {
        if (hc[i] != c[i]) {
            printf("[%d] The result is not matched! (%d, %d)\n", i, hc[i], c[i]);
            result = false;
        }
    }

    if (result)
        printf("GPU works well!\n");

    timer.printTimer();

    // Release host memory
    delete[] a; delete[] b; delete[] c;

    return 0;
}